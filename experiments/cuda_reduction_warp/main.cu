
#include <hip/hip_runtime.h>
#include <cstdio>



__global__ void reduce(int * data) {

    __shared__ volatile int shmem[64]; // VOLATILE !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    // there will be exactly 32 threads = 1 warp

    int tid = threadIdx.x;
    shmem[tid] = data[tid];
    shmem[tid + 32] = data[tid + 32];
    
    shmem[tid] += shmem[tid + 32];
    shmem[tid] += shmem[tid + 16];
    shmem[tid] += shmem[tid +  8];
    shmem[tid] += shmem[tid +  4];
    shmem[tid] += shmem[tid +  2];
    shmem[tid] += shmem[tid +  1];

    if(tid == 0)
        data[0] = shmem[0];
}


int main() {

    int size = 64;

    int * x;
    hipHostMalloc(&x, size * sizeof(*x), hipHostMallocDefault);
    for(int i = 0; i < size; i++)
        x[i] = rand() % 100;
    
    // for(int i = 0; i < size; i++)
    //     printf("%2d:%2d\n", i, (int)x[i]);

    int resultCpu = 0;
    for(int i = 0; i < 64; i++)
    resultCpu += x[i];
    printf("CPU result: %d\n", (int)resultCpu);
    

    int * d_x;
    hipMalloc(&d_x, size * sizeof(*d_x));
    hipMemcpy(d_x, x, size * sizeof(*x), hipMemcpyHostToDevice);
    reduce<<< 1, 32 >>>(d_x);
    hipDeviceSynchronize();
    int resultGpu;
    hipMemcpy(&resultGpu, d_x, sizeof(*x), hipMemcpyDeviceToHost);
    printf("GPU result: %d\n", (int)resultGpu);


    hipFree(d_x);
    hipHostFree(x);


    return 0;
}

