#include "hip/hip_runtime.h"
#include "daxpy.h"


__global__ void d_daxpy(double alpha, double * d_x, double * d_y, long long count)
{
    long long index = blockIdx.x * blockDim.x + threadIdx.x;

    for (long long i = index; i < count; i += gridDim.x)
    {
        d_y[i] += alpha * d_x[i];
    }
}



void daxpy(double alpha, double * x, double * y, long long count)
{
    long long bytes = count * sizeof(*x);

    double *d_x, *d_y;
    hipMalloc(&d_x, bytes);
    hipMalloc(&d_y, bytes);

    hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);

    d_daxpy<<< 8, 256 >>>(alpha, d_x, d_y, count);

    hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}




